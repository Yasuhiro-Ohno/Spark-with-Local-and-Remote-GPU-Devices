
#include <hip/hip_runtime.h>
extern "C"
#define BLOCKSIZE 128

__global__ void sum(double *d_A, double *d_B)
{
	__shared__ double sdata[BLOCKSIZE];

	unsigned int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;
	sdata[tid] = d_A[id];
	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) {
		d_B[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
	}
}
