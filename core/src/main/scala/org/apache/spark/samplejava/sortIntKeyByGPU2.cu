
#include <hip/hip_runtime.h>
extern "C"

__global__ void sortkey(int *ks_rdd_array, int *cp_rdd_array, int *num_array, int j, int k, int count)
{
	const unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if (count == 0) {
		num_array[i] = i;
		cp_rdd_array[i] = ks_rdd_array[i];
		__syncthreads();
	}
	unsigned int ixj = i^j;

	if ((ixj)>i) {
		if ((i&k)==0) {
			if (cp_rdd_array[i] < cp_rdd_array[ixj]) {
				int tmp = cp_rdd_array[i];
				cp_rdd_array[i] = cp_rdd_array[ixj];
				cp_rdd_array[ixj] = tmp;

				int tmp2 = num_array[i];
				num_array[i] = num_array[ixj];
				num_array[ixj] = tmp2;
				__syncthreads();
			}
		} else {
			if (cp_rdd_array[i] > cp_rdd_array[ixj]) {
				int tmp = cp_rdd_array[i];
				cp_rdd_array[i] = cp_rdd_array[ixj];
				cp_rdd_array[ixj] = tmp;

				int tmp2 = num_array[i];
				num_array[i] = num_array[ixj];
				num_array[ixj] = tmp2;
				__syncthreads();
			}
		}
	}
}
