
#include <hip/hip_runtime.h>
extern "C"
#define BLOCKSIZE 128

__global__ void wordconversion(char *rdd_array, char *result_array, int f)
{
	unsigned int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	//unsigned int tid = threadIdx.x;
	if (f == 0) {
		if (rdd_array[id]>='a' && rdd_array[id]<='z'){
			result_array[id] = rdd_array[id]-32;
		} else {
			result_array[id] = rdd_array[id];
		}
	} else {
		if (rdd_array[id]>='A' && rdd_array[id]<='Z'){
			result_array[id] = rdd_array[id]+32;
		} else {
			result_array[id] = rdd_array[id];
		}
	}
}
