
#include <hip/hip_runtime.h>
extern "C"
#define BLOCKSIZE 128

__global__ void sortkey(int *ks_rdd_array, int *cp_rdd_array, int *num_array, int j, int k, int count, int loop)
{
	const unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	int s = loop*32768*BLOCKSIZE;
//	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (count == 0) {
		num_array[i + s] = i;
		cp_rdd_array[i + s] = ks_rdd_array[i + s];
		__syncthreads();
	}
	unsigned int ixj = i^j;

	if ((ixj)>i) {
		if ((i&k)==0) {
			if (cp_rdd_array[i + s] > cp_rdd_array[ixj + s]) {
				int tmp = cp_rdd_array[i + s];
				cp_rdd_array[i + s] = cp_rdd_array[ixj + s];
				cp_rdd_array[ixj + s] = tmp;

				int tmp2 = num_array[i + s];
				num_array[i + s] = num_array[ixj + s];
				num_array[ixj + s] = tmp2;
				__syncthreads();
			}
		} else {
			if (cp_rdd_array[i + s] < cp_rdd_array[ixj + s]) {
				int tmp = cp_rdd_array[i + s];
				cp_rdd_array[i + s] = cp_rdd_array[ixj + s];
				cp_rdd_array[ixj + s] = tmp;

				int tmp2 = num_array[i + s];
				num_array[i + s] = num_array[ixj + s];
				num_array[ixj + s] = tmp2;
				__syncthreads();
			}
		}
	}
}
