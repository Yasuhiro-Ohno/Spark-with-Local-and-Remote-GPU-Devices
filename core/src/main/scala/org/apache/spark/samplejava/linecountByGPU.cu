
#include <hip/hip_runtime.h>
extern "C"
#define BLOCKSIZE 128

__global__ void linecount(char *rdd_array, int *result_array)
{
	__shared__ int sdata[BLOCKSIZE];

	unsigned int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;
	if (rdd_array[id] == '\\'){
		sdata[tid] = 1;
	} else {
		sdata[tid] = 0;
	}
	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) {
			sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    result_array[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
  }
}
