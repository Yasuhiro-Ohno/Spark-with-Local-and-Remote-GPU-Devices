
#include <hip/hip_runtime.h>
extern "C"
#define BLOCKSIZE 128

__global__ void reduce(int *rdd_array, int *result_array)
{
	__shared__ int sdata[BLOCKSIZE];

	unsigned int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;
	sdata[tid] = rdd_array[id];
	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) {
		result_array[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
	}
}
