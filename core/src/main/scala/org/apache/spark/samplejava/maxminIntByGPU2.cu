
#include <hip/hip_runtime.h>
extern "C"
#define BLOCKSIZE 128
__global__ void maxmin2(int *rdd_array, int *max_array){
	unsigned int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	max_array[id] = rdd_array[id];
}
