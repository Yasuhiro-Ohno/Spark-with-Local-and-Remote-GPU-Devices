
#include <hip/hip_runtime.h>
extern "C"

//#include <stdio.h>
//#include <stdlib.h>

__global__ void sort(int *rdd_array, int j, int k, int f)
{
	unsigned int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	unsigned int ixj = i^j;

	if (f == 0) {//ascending order
		if ((ixj)>i) {
			if ((i&k)==0) {
				if (rdd_array[i] > rdd_array[ixj]) {
					int tmp = rdd_array[i];
					rdd_array[i] = rdd_array[ixj];
					rdd_array[ixj] = tmp;
				}
			} else {
//			if ((i&k)!=0) {
				if (rdd_array[i] < rdd_array[ixj]) {
					int tmp = rdd_array[i];
					rdd_array[i] = rdd_array[ixj];
					rdd_array[ixj] = tmp;
				}
			}
		}
	} else {//descending order
		if ((ixj)>i) {
			if ((i&k)==0) {
				if (rdd_array[i] < rdd_array[ixj]) {
					int tmp = rdd_array[i];
					rdd_array[i] = rdd_array[ixj];
					rdd_array[ixj] = tmp;
				}
			} else {
//			if ((i&k)!=0) {
				if (rdd_array[i] > rdd_array[ixj]) {
					int tmp = rdd_array[i];
					rdd_array[i] = rdd_array[ixj];
					rdd_array[ixj] = tmp;
				}
			}
		}
	}
}
