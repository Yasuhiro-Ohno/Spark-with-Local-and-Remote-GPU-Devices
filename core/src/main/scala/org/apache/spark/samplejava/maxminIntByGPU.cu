
#include <hip/hip_runtime.h>
extern "C"
#define BLOCKSIZE 128

__global__ void maxmin(int *rdd_array, int *max_array, int len)
{
	unsigned int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
//	unsigned int tid = threadIdx.x;

	/*
	for (unsigned int s = 1; s <= rep; s++) {
		if (id % 2 == 0) {
			if(max_array[id]>max_array[id+1]) {
				max_array2[id/2] = max_array[id];
				max_array2[id/2+len/2] = max_array[id+1];
			} else {
				max_array2[id/2] = max_array[id+1];
				max_array2[id/2+len/2] = max_array[id];
			}
		}
		__syncthreads();
		max_array[id] = max_array2[id];
		__syncthreads();
	}
	*/


//	for (unsigned int s = 1; s <= rep; s++) {
		if (id < len/2) {
			int v1 = max_array[id*2];
			int v2 = max_array[id*2+1];
			if (v1 > v2) {
				rdd_array[id] = v1;
				rdd_array[id + len/2] = v2;
			} else {
				rdd_array[id] = v2;
				rdd_array[id + len/2] = v1;
			}
//			__syncthreads();
		}
//		max_array[id] = max_array2[id];
//		__syncthreads();
//	}
//	__syncthreads();

//	if (id == 0) {
//		result_array[0] = max_array[0];
//		result_array[1] = max_array[len-1];
//	}
}
