
#include <hip/hip_runtime.h>
extern "C"

//#include <stdio.h>
//#include <stdlib.h>

__global__ void sort(int *rdd_array, int DataSize)
{
	unsigned int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	for (unsigned int length = 2; length <= DataSize; length = length * 2) {

		for (unsigned int mlength = length/2; mlength > 0; mlength = mlength/2) {
			unsigned int ixj = id ^ mlength;

			if (ixj > id) {
				if ((id & length) == 0) {
					if (rdd_array[id] > rdd_array[ixj]) {
						int tmp = rdd_array[ixj];
						rdd_array[ixj] = rdd_array[id];
						rdd_array[id] = tmp;
					}
				} else {
					if (rdd_array[id] < rdd_array[ixj]) {
						int tmp = rdd_array[ixj];
						rdd_array[ixj] = rdd_array[id];
						rdd_array[id] = tmp;
					}
				}
			}
			__threadfence();
			__syncthreads();
		}
	}
}
