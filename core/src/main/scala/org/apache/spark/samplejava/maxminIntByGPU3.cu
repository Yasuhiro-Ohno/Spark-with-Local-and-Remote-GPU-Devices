
#include <hip/hip_runtime.h>
extern "C"
#define BLOCKSIZE 128
__global__ void maxmin3(int *rdd_array, int *result_array, int len){
	unsigned int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if (id == 0){
		result_array[0] = rdd_array[0];
//		result_array[1] = rdd_array[len-1];
	}
	if (id == len-1){
		result_array[1] = rdd_array[len-1];
	}
}
