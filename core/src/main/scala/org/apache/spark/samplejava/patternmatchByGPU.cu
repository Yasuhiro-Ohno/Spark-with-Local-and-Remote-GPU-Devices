
#include <hip/hip_runtime.h>
extern "C"
#define BLOCKSIZE 128

//__global__ void patternmatch(char *rdd_array, char *pattern_array, char *flag_array, char *result_array, int *count_array, int length, int p_length)
__global__ void patternmatch(char *rdd_array, char *pattern_array, char *flag_array, int length, int p_length)
{
	unsigned int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	//unsigned int tid = threadIdx.x;

	flag_array[id] = '0';
	//result_array[id] = '0';
	__syncthreads();

	int i = 0;
	
	if (p_length == 1) {//p_length == 1
		if (rdd_array[id] == pattern_array[0]) {
			flag_array[id] = rdd_array[id];
		}
	} else if (p_length == 2) {//p_length == 2
		if (rdd_array[id] == pattern_array[0] && 
					rdd_array[id+1] == pattern_array[1]) {
			flag_array[id] = rdd_array[id];
			flag_array[id+1] = rdd_array[id+1];
		}
	} else if (p_length == 3) {//p_length == 3
		if (rdd_array[id] == pattern_array[0] &&
					rdd_array[id+1] == pattern_array[1] &&
						rdd_array[id+2] == pattern_array[2]) {
			flag_array[id] = rdd_array[id];
			flag_array[id+1] = rdd_array[id+1];
			flag_array[id+2] = rdd_array[id+2];
		}
	} else if (p_length == 4) {//p_length == 4
		if (rdd_array[id] == pattern_array[0] &&
					rdd_array[id+1] == pattern_array[1] &&
						rdd_array[id+2] == pattern_array[2] &&
							rdd_array[id+3] == pattern_array[3]) {
			flag_array[id] = rdd_array[id];
			flag_array[id+1] = rdd_array[id+1];
			flag_array[id+2] = rdd_array[id+2];
			flag_array[id+3] = rdd_array[id+3];
		}
	} else if (p_length == 5) {//p_length == 5
		if (rdd_array[id] == pattern_array[0] &&
					rdd_array[id+1] == pattern_array[1] &&
						rdd_array[id+2] == pattern_array[2] &&
							rdd_array[id+3] == pattern_array[3] &&
								rdd_array[id+4] == pattern_array[4]) {
			flag_array[id] = rdd_array[id];
			flag_array[id+1] = rdd_array[id+1];
			flag_array[id+2] = rdd_array[id+2];
			flag_array[id+3] = rdd_array[id+3];
			flag_array[id+4] = rdd_array[id+4];
	//	}
	//}
	//__syncthreads();

			
			i = 0;
			while ((rdd_array[id-i] != '\\')) {
				if (id-i == 0) {
					break;
				}
				flag_array[id-i] = rdd_array[id-i];
				i++;
			}

			i = 4;
			do {
				i++;
				if (id+i >= length-1) {
					break;
				}
				flag_array[id+i] = rdd_array[id+i];
			} while (rdd_array[id+i] != '\\');
			

		}
	}
	__syncthreads();

	/*
  if (id == 0) {
		int count = 0;
		for (int j = 0; j < length; j++) {
			if (flag_array[j] != '0') {
				result_array[count] = flag_array[j];
				count++;
			}
			//__syncthreads();
		}
		//__syncthreads();
		count_array[0] = count;
  }
	//__syncthreads();
	*/
}
