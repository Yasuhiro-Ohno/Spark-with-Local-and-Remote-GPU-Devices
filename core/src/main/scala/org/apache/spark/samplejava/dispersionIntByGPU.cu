
#include <hip/hip_runtime.h>
extern "C"
#define BLOCKSIZE 128

__global__ void dispersion(int *rdd_array, double *result_array2, double avg)
{
	__shared__ double sdata[BLOCKSIZE];

	unsigned int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;
	sdata[tid] = (rdd_array[id] - avg) * (rdd_array[id] - avg);
	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) {
		result_array2[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
	}
}
